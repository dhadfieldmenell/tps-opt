#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "cdist.cuh"


__global__ void sqdistKernel(float* x, float* y, int dim, float* z) {
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  if (ix < dim * dim) {
    int x_ix = ix / dim;
    int y_ix = ix - x_ix * dim;
    float diff = x[x_ix] - y[y_ix];
    z[ix] = diff * diff;
  }
}

float* sqdistWrapper(float* x, float* y, int xdim, int ydim)
{
  int x_size = xdim * sizeof(float);
  int y_size = ydim * sizeof(float);
  int z_size = x_size * y_size / sizeof(float);

  float* x_gpu = 0;
  float* y_gpu = 0;
  float* z_gpu = 0;

  hipError_t err_x = hipMalloc((void **) &x_gpu, x_size);
  hipError_t err_y = hipMalloc((void **) &y_gpu, y_size);
  hipError_t err_z = hipMalloc((void **) &z_gpu, z_size);

  if ((err_x != hipSuccess) ||
      (err_y != hipSuccess) ||
      (err_z != hipSuccess))
    {
      if (x_gpu) hipFree(x_gpu);
      if (y_gpu) hipFree(y_gpu);
      if (z_gpu) hipFree(z_gpu);
      fprintf(stderr, "!!!! GPU memory allocation error\n");
      return 0;
    }

  err_x = hipMemcpy(x_gpu, x, x_size, hipMemcpyHostToDevice);
  err_y = hipMemcpy(y_gpu, y, y_size, hipMemcpyHostToDevice);  

  if ((err_x != hipSuccess) ||
      (err_y != hipSuccess))
    {
      if (x_gpu) hipFree(x_gpu);
      if (y_gpu) hipFree(y_gpu);
      if (z_gpu) hipFree(z_gpu);
      fprintf(stderr, "!!!! GPU memory allocation error\n");
      return 0;
    }

  sqdistKernel<<<xdim, ydim>>>(x_gpu, y_gpu, xdim, z_gpu);
  
  float* z = new float[xdim*ydim];
  err_z = hipMemcpy(z, z_gpu, z_size, hipMemcpyDeviceToHost);

  if (x_gpu) hipFree(x_gpu);
  if (y_gpu) hipFree(y_gpu);
  if (z_gpu) hipFree(z_gpu);

  return z;
}

int main(void)
{
  float* x = new float[10];
  for(int i = 0; i < 10; ++i)
    x[i] = (float) i;

  float* z = sqdistWrapper(x, x, 10, 10);
  for(int i = 0; i < 10; ++i){
    for(int j = 0; j < 10; ++j)
      printf("z[%i, %i] = %2f", i, j, z[i*10 + j]);
    printf("\n");
  }

  free(x);
  delete[] z;
  
  return 0;
}
